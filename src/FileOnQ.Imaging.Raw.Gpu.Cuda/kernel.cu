#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "ImagingGpu.h"

hipError_t proccessBitmapWithCuda(unsigned char* bitmap, unsigned char* data, unsigned int bitmapSize, unsigned int size, unsigned int width, unsigned height, int* error);

//https://developer.nvidia.com/blog/even-easier-introduction-cuda/

__global__ void process_bitmap_kernel(unsigned char* bitmap, unsigned char* data, int pixels, int imageStride, int offset)
{
	// This just works, do not touch
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int gpuStride = blockDim.x * gridDim.x;

	// As we iterate through pixel space which is different than buffer space (3 bytes for 1 pixel).
	// We need to ensure when i == number of pixels we set the last offset items in the bitmap to 0
	for (int i = index; i <= pixels; i+= gpuStride)
	{
		// Calculate the input pointer position
		int position = index * 3;
	
		// Calculate the row on the output bitmap pointer
		// Consider
		//  position = 20226
		//  imageStride = 20226
		//  20226 / 20226 = 1, but we want it to equal 0 or we will be off by 1
		// We need to subtract 1 to ensure we are on the correct row
		int row = (position - 1) / imageStride;
	
		// calculate the position of the output bitmap pointer
		int bitmapPosition = position + (row * offset);

		// At the end of every row, calcuated by the input pointer,
		// we need to set the offset to 0 or empty data.
		if (position > 0 && position % imageStride == 0)
		{
			for (int j = 0; j < offset; j++)
			{
				bitmap[bitmapPosition] =  0;

				// Update the bitmap position as we added 2 bytes per row
				bitmapPosition++;
			}
		}

		// Converting a PPM bitmap to a BMP bitmap flips
		// the red and blue value
		bitmap[bitmapPosition + 2] = data[position];
		bitmap[bitmapPosition + 1] = data[position + 1];
		bitmap[bitmapPosition] = data[position + 2];
	}
}

unsigned char* process_bitmap(unsigned char* data, int size, int width, int height, int* length, int* error)
{
	int offset = height * (width % 4);
	int bitmapSize = size + offset;
	*length = bitmapSize;
	unsigned char* bitmap = new unsigned char[bitmapSize];

	hipError_t cudaStatus = proccessBitmapWithCuda(bitmap, data, bitmapSize, size, width, height, error);
	if (cudaStatus != hipSuccess) {
		bitmap[0] = 1;
		fprintf(stderr, hipGetErrorString(cudaStatus));
	}

	return bitmap;
}

// Error codes
// -1 = No CUDA-Capable GPU
// -2 = Unable to allocate GPU buffers
// -3 = Unable to copy host memory to device memory
// -4 = Unable to launch CUDA kernel
// -5 = Error while running CUDA kernel
// -6 = Unable to copy device memory to host memory
hipError_t proccessBitmapWithCuda(unsigned char* bitmap, unsigned char* data, unsigned int bitmapSize, unsigned int size, unsigned int width, unsigned int height, int* error)
{
	unsigned char* dev_bitmap = 0;
	unsigned char* dev_data = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		*error = -1;
		goto Error;
	}

	// Allocate GPU buffers for intpu data and output bitmap
	cudaStatus = hipMalloc((void**)&dev_bitmap, bitmapSize * sizeof(unsigned char));
	if (cudaStatus != hipSuccess) {
		*error = -2;
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_data, size * sizeof(unsigned char));
	if (cudaStatus != hipSuccess) {
		*error = -2;
		goto Error;
	}

	// Copy input image from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_bitmap, bitmap, bitmapSize * sizeof(unsigned char), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		*error = -3;
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_data, data, size * sizeof(unsigned char), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		*error = -3;
		goto Error;
	}

	// TODO - determine best block size and thread count. passing size is too large of a value

	
	int pixels = size / 3;
	int blockSize = 256;
	int numberOfBlocks = (pixels + blockSize - 1) / blockSize;

	int offset = width % 4;
	int stride = width * 3;
	
	// Launch a kernel on the GPU with one thread for each element.
	process_bitmap_kernel<<<numberOfBlocks, blockSize>>>(dev_bitmap, dev_data, pixels, stride, offset);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		*error = -4;
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		*error = -5;
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(bitmap, dev_bitmap, bitmapSize * sizeof(unsigned char), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		*error = -6;
		goto Error;
	}

Error:
	hipFree(dev_bitmap);
	hipFree(dev_data);

	return cudaStatus;
}

bool is_cuda_capable()
{
	int c = 0;
	int* count = &c;
	hipError_t status = hipGetDeviceCount(count);
	if (status != hipSuccess)
		return false;
	
	return *count > 0;
}