#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "ImagingGpu.h"

hipError_t proccessBitmapWithCuda(unsigned char* bitmap, unsigned char* data, unsigned int bitmapSize, unsigned int size, unsigned int width, unsigned height, int* error);

//https://developer.nvidia.com/blog/even-easier-introduction-cuda/

__global__ void process_bitmap_kernel(unsigned char* bitmap, unsigned char* data, int pixels, int imageStride, int offset)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int gpuStride = blockDim.x * gridDim.x;

	int position = index * 3;

	int row = position / imageStride;
	int bitmapPosition = (index * 3) + (row * offset);

	for (int i = index; i < pixels; i+= gpuStride)
	{
		if (position > 0 && position % imageStride == 0)
		{
			for (int j = 0; j < offset; j++)
			{
				bitmap[bitmapPosition] = 0;
				bitmapPosition++;
			}
		}

		bitmap[bitmapPosition + 2] = data[position];
		bitmap[bitmapPosition + 1] = data[position + 1];
		bitmap[bitmapPosition] = data[position + 2];
		
		position += 3;
		bitmapPosition += 3;
	}
}

// used for debugging, we should move this into a standalone console app
//int main()
//{
//	unsigned char* data = new unsigned char[9];
//	data[0] = 1;
//	data[1] = 2;
//	data[2] = 3;
//	data[3] = 4;
//	data[4] = 5;
//	data[5] = 6;
//	data[6] = 7;
//	data[7] = 8;
//	data[8] = 9;
//
//	unsigned char* bitmap = process_bitmap(data, 9);
//
//	/*int* a = new int[2];
//	int* b = new int[2];
//	int* c = new int[2];
//	a[0] = 5;
//	b[0] = 3;
//	a[1] = 2;
//	b[1] = 2;
//
//	addWithCuda(c, a, b, 2);*/
//	return 0;
//}

unsigned char* process_bitmap(unsigned char* data, int size, int width, int height, int* error)
{
	int offset = height * (width % 4);
	int bitmapSize = size + offset;
	unsigned char* bitmap = new unsigned char[bitmapSize];

	hipError_t cudaStatus = proccessBitmapWithCuda(bitmap, data, bitmapSize, size, width, height, error);
	if (cudaStatus != hipSuccess) {
		bitmap[0] = 1;
		fprintf(stderr, hipGetErrorString(cudaStatus));
	}

	return bitmap;
}

// Error codes
// -1 = No CUDA-Capable GPU
// -2 = Unable to allocate GPU buffers
// -3 = Unable to copy host memory to device memory
// -4 = Unable to launch CUDA kernel
// -5 = Error while running CUDA kernel
// -6 = Unable to copy device memory to host memory
hipError_t proccessBitmapWithCuda(unsigned char* bitmap, unsigned char* data, unsigned int bitmapSize, unsigned int size, unsigned int width, unsigned int height, int* error)
{
	unsigned char* dev_bitmap = 0;
	unsigned char* dev_data = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		*error = -1;
		goto Error;
	}

	// Allocate GPU buffers for intpu data and output bitmap
	cudaStatus = hipMalloc((void**)&dev_bitmap, bitmapSize * sizeof(unsigned char));
	if (cudaStatus != hipSuccess) {
		*error = -2;
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_data, size * sizeof(unsigned char));
	if (cudaStatus != hipSuccess) {
		*error = -2;
		goto Error;
	}

	// Copy input image from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_bitmap, bitmap, bitmapSize * sizeof(unsigned char), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		*error = -3;
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_data, data, size * sizeof(unsigned char), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		*error = -3;
		goto Error;
	}

	// TODO - determine best block size and thread count. passing size is too large of a value

	
	int pixels = size / 3;
	int blockSize = 256;
	int numberOfBlocks = (pixels + blockSize - 1) / blockSize;

	int offset = width % 4;
	int stride = width * 3;
	
	// Launch a kernel on the GPU with one thread for each element.
	process_bitmap_kernel<<<numberOfBlocks, blockSize>>>(dev_bitmap, dev_data, pixels, stride, offset);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		*error = -4;
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		*error = -5;
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(bitmap, dev_bitmap, bitmapSize * sizeof(unsigned char), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		*error = -6;
		goto Error;
	}

Error:
	hipFree(dev_bitmap);
	hipFree(dev_data);

	return cudaStatus;
}